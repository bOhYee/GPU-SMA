#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <sys/stat.h>

#include "../inc/constants.h"
#include "../inc/smatch.h"

/* Prototypes
*/
int read_text (char *file_path, unsigned char *storage);
void cpu_call (int algorithm, unsigned char *text, int text_size, unsigned char *pattern, int pattern_size, int *results);
void gpu_call (int algorithm, unsigned char *text, int text_size, unsigned char *pattern, int pattern_size, int *results);

int main (int argc, char *argv[]) {

    int text_size, pattern_size, matches;
    int *results;
    unsigned char *text;
    unsigned char *pattern;
    unsigned char stat_pattern[] = "ipsum";

    struct stat text_info, pattern_info;

    // Verify parameters
    if (argc < 2 && argc > 3) {
        fprintf(stderr, "Call to the program performed incorrectly!\n");
        fprintf(stderr, "Send only two arguments at maximum: the path for the text to be searched on and the path for the pattern to use\n");
        exit(EXIT_FAILURE);
    }

    if (stat(argv[1], &text_info) == -1) {
        fprintf(stderr, "Indicated path for text might not be correct\n");
        exit(EXIT_FAILURE);
    }

    text_size = text_info.st_size;
    text = (unsigned char*) malloc(text_size * sizeof(unsigned char));

    if (argc == 3) {
        if (stat(argv[2], &pattern_info) == -1) {
            fprintf(stderr, "Indicated path for pattern might not be correct\n");
            exit(EXIT_FAILURE);
        }
    
        pattern_size = pattern_info.st_size;
        pattern = (unsigned char *) malloc(pattern_size * sizeof(unsigned char));    
    }
    else {
        pattern_size = strlen((char *) stat_pattern);
        pattern = stat_pattern;
    }

    results = (int*) malloc((text_size - pattern_size) * sizeof(int));

    if (text == NULL || pattern == NULL || results == NULL) {
        fprintf(stderr, "Error during memory allocation!\n");
        exit(EXIT_FAILURE);
    }
    
    if(read_text(argv[1], text) == 1) {
        fprintf(stderr, "Error during file reading!\n");
        exit(EXIT_FAILURE);
    }

    if(argc == 3 && read_text(argv[2], pattern) == 1) {
        fprintf(stderr, "Error during file reading!\n");
        exit(EXIT_FAILURE);
    }

    printf("%d %d", text_size, pattern_size);
    //cpu_call(1, text, text_size, pattern, pattern_size, results);
    gpu_call(1, text, text_size, pattern, pattern_size, results);

    // Results
    matches = 0;
    for (int i = 0; i < (text_size - pattern_size); i++){
        if (results[i] == 1){
            matches++;
            printf("Match found at index: %d\n", i+1);
        }
    }
    printf("Total matches: %d\n", matches);

    // Release memory
    free(text);
    return 0;
}

// Copy the text from the file inside the memory
int read_text (char *file_path, unsigned char *storage) {

    int i;
    FILE *in_file;

    in_file = fopen(file_path, "r");
    if (in_file == NULL) 
        return 1;

    i = 0;
    while (fscanf(in_file, "%c", &storage[i++]) > 0);

    return 0;
}

void cpu_call (int algorithm, unsigned char *text, int text_size, unsigned char *pattern, int pattern_size, int *results) {

    double diff;
    time_t start, end;

    printf("Launching the algorithm on the host device (CPU)...\n");
    time(&start);

    switch (algorithm) {
        case NAIVE_RK:
        default:
            rk_cpu(text, text_size, pattern, pattern_size, results);
            break;
    }

    time(&end);
    diff = difftime(end, start);
    printf("Operations terminated in %lf seconds.\n", diff);

}

void gpu_call (int algorithm, unsigned char *text, int text_size, unsigned char *pattern, int pattern_size, int *results) {

    int grid_size_x, grid_size_y, block_size_x, block_size_y;
    int *gpu_results;
    float elaboration_time;
    unsigned char *gpu_text, *gpu_pattern;
    hipEvent_t start, end;

    // Kernel parameters definition
    printf("Defining grid and block dimensions...\n");
    switch (algorithm) {
        case NAIVE_RK:
        default:
            block_size_x = BLOCK_DIMENSION;
            block_size_y = BLOCK_DIMENSION;

            grid_size_x = ceil(sqrt(text_size / (block_size_x * block_size_y))) + 1;
            grid_size_y = grid_size_x;
            break;
    }

    dim3 gridDimension(grid_size_x, grid_size_y);
    dim3 blockDimension(block_size_x, block_size_y);
    printf("Grid: %dx%d\nBlocks: %dx%d\n", grid_size_x, grid_size_y, block_size_x, block_size_y);

    // Events
    hipEventCreate(&start);
    hipEventCreate(&end);

    // GPU allocations and copy
    hipMalloc((void **) &gpu_text, text_size * sizeof(unsigned char));
    hipMalloc((void **) &gpu_pattern, pattern_size * sizeof(unsigned char));
    hipMemcpy(gpu_text, text, text_size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(gpu_pattern, pattern, pattern_size * sizeof(unsigned char), hipMemcpyHostToDevice);

    printf("Launching the kernel...\n");
    hipEventRecord(start);
    switch (algorithm) {
        case NAIVE_RK:
        default:
            hipMalloc((void **) &gpu_results, (text_size-pattern_size) * sizeof(int));
            naive_rk_gpu<<<gridDimension, blockDimension>>>(gpu_text, text_size, gpu_pattern, pattern_size, ceil(text_size / SUBTEXTS_NUM), gpu_results);
            hipMemcpy(results, gpu_results, (text_size-pattern_size) * sizeof(int), hipMemcpyDeviceToHost);
            break;
    }
    hipEventRecord(end);

    // Compute time for elaboration
    hipEventElapsedTime(&elaboration_time, start, end);
    //elaboration_time /= 1000;
    printf("Kernel operations terminated in %f seconds\n", elaboration_time);
}